#include "hip/hip_runtime.h"
// This software is in the public domain. Where that dedication is not
// recognized, you are granted a perpetual, irrevocable license to copy
// and modify this file as you see fit.

#include "ebsynth.h"
#include "patchmatch_gpu.h"

#define FOR(A,X,Y) for(int Y=0;Y<A.height();Y++) for(int X=0;X<A.width();X++)

A2V2i nnfInitRandom(const V2i& targetSize,
                    const V2i& sourceSize,
                    const int  patchSize)
{
  A2V2i NNF(targetSize);
  const int r = patchSize/2;

  for (int i = 0; i < NNF.numel(); i++)
  {
      NNF[i] = V2i
      (
          r+(rand()%(sourceSize[0]-2*r)),
          r+(rand()%(sourceSize[1]-2*r))
      );
  }

  return NNF;
}

A2V2i nnfUpscale(const A2V2i& NNF,
                 const int    patchSize,
                 const V2i&   targetSize,
                 const V2i&   sourceSize)
{
  A2V2i NNF2x(targetSize);

  FOR(NNF2x,x,y)
  {
    NNF2x(x,y) = NNF(clamp(x/2,0,NNF.width()-1),
                     clamp(y/2,0,NNF.height()-1))*2+V2i(x%2,y%2);
  }

  FOR(NNF2x,x,y)
  {
    const V2i nn = NNF2x(x,y);

    NNF2x(x,y) = V2i(clamp(nn(0),patchSize,sourceSize(0)-patchSize-1),
                     clamp(nn(1),patchSize,sourceSize(1)-patchSize-1));
  }

  return NNF2x;
}

template<int N, typename T, int M>
__global__ void krnlVotePlain(      TexArray2<N,T,M> target,
                              const TexArray2<N,T,M> source,
                              const TexArray2<2,int> NNF,
                              const int              patchSize)
{
  const int x = blockDim.x*blockIdx.x + threadIdx.x;
  const int y = blockDim.y*blockIdx.y + threadIdx.y;

  if (x<target.width && y<target.height)
  {
    const int r = patchSize / 2;

    Vec<N,float> sumColor = zero<Vec<N,float>>::value();
    float sumWeight = 0;

    for (int py = -r; py <= +r; py++)
    for (int px = -r; px <= +r; px++)
    {
      /*
      if
      (
        x+px >= 0 && x+px < NNF.width () &&
        y+py >= 0 && y+py < NNF.height()
      )
      */
      {
        const V2i n = NNF(x+px,y+py)-V2i(px,py);

        /*if
        (
          n[0] >= 0 && n[0] < S.width () &&
          n[1] >= 0 && n[1] < S.height()
        )*/
        {
          const float weight = 1.0f;
          sumColor += weight*Vec<N,float>(source(n(0),n(1)));
          sumWeight += weight;
        }
      }
    }

    const Vec<N,T> v = Vec<N,T>(sumColor/sumWeight);
    target.write(x,y,v);
  }
}

template<int N, typename T, int M>
__global__ void krnlVoteWeighted(      TexArray2<N,T,M>   target,
                                 const TexArray2<N,T,M>   source,
                                 const TexArray2<2,int>   NNF,
                                 const TexArray2<1,float> E,
                                 const int patchSize)
{
  const int x = blockDim.x*blockIdx.x + threadIdx.x;
  const int y = blockDim.y*blockIdx.y + threadIdx.y;

  if (x<target.width && y<target.height)
  {
    const int r = patchSize / 2;

    Vec<N,float> sumColor = zero<Vec<N,float>>::value();
    float sumWeight = 0;

    for (int py = -r; py <= +r; py++)
    for (int px = -r; px <= +r; px++)
    {
      /*
      if
      (
        x+px >= 0 && x+px < NNF.width () &&
        y+py >= 0 && y+py < NNF.height()
      )
      */
      {
        const V2i n = NNF(x+px,y+py)-V2i(px,py);

        /*if
        (
          n[0] >= 0 && n[0] < S.width () &&
          n[1] >= 0 && n[1] < S.height()
        )*/
        {
          const float error = E(x+px,y+py)(0)/(patchSize*patchSize*N);
          const float weight = 1.0f/(1.0f+error);
          sumColor += weight*Vec<N,float>(source(n(0),n(1)));
          sumWeight += weight;
        }
      }
    }

    const Vec<N,T> v = Vec<N,T>(sumColor/sumWeight);
    target.write(x,y,v);
  }
}

template<int N, typename T, int M>
__device__ Vec<N,T> sampleBilinear(const TexArray2<N,T,M>& I,float x,float y)
{
  const int ix = x;
  const int iy = y;

  const float s = x-ix;
  const float t = y-iy;

  // XXX: clamp!!!
  return Vec<N,T>((1.0f-s)*(1.0f-t)*Vec<N,float>(I(ix  ,iy  ))+
                  (     s)*(1.0f-t)*Vec<N,float>(I(ix+1,iy  ))+
                  (1.0f-s)*(     t)*Vec<N,float>(I(ix  ,iy+1))+
                  (     s)*(     t)*Vec<N,float>(I(ix+1,iy+1)));
};

template<int N, typename T, int M>
__global__ void krnlResampleBilinear(TexArray2<N,T,M> O,
                                     const TexArray2<N,T,M> I)
{
  const int x = blockDim.x*blockIdx.x + threadIdx.x;
  const int y = blockDim.y*blockIdx.y + threadIdx.y;

  if (x<O.width && y<O.height)
  {
    const float s = float(I.width)/float(O.width);
    O.write(x,y,sampleBilinear(I,s*float(x),s*float(y)));
  }
}

template<int N, typename T, int M>
__global__ void krnlEvalMask(      TexArray2<1,unsigned char> mask,
                             const TexArray2<N,T,M> style,
                             const TexArray2<N,T,M> style2,
                             const int stopThreshold)
{
  const int x = blockDim.x*blockIdx.x + threadIdx.x;
  const int y = blockDim.y*blockIdx.y + threadIdx.y;

  if (x<mask.width && y<mask.height)
  {
    const Vec<N,T> s  = style(x,y);
    const Vec<N,T> s2 = style2(x,y);

    int maxDiff = 0;
    for(int c=0;c<N;c++)
    {
      const int diff = std::abs(int(s[c])-int(s2[c]));
      maxDiff = diff>maxDiff ? diff:maxDiff;
    }

    const Vec<1,unsigned char> msk = maxDiff < stopThreshold ? Vec<1,unsigned char>(0) : Vec<1,unsigned char>(255);

    mask.write(x,y,msk);
  }
}

__global__ void krnlDilateMask(TexArray2<1,unsigned char> mask2,
                               const TexArray2<1,unsigned char> mask,
                               const int patchSize)
{
  const int x = blockDim.x*blockIdx.x + threadIdx.x;
  const int y = blockDim.y*blockIdx.y + threadIdx.y;

  if (x<mask.width && y<mask.height)
  {
    const int r = patchSize / 2;

    Vec<1,unsigned char> msk = Vec<1,unsigned char>(0);

    for (int py = -r; py <= +r; py++)
    for (int px = -r; px <= +r; px++)
    {
      if (mask(x+px,y+py)[0]==255) { msk = Vec<1,unsigned char>(255); }
    }

    mask2.write(x,y,msk);
  }
}

template<int N, typename T, int M>
void resampleGPU(      TexArray2<N,T,M>& O,
                 const TexArray2<N,T,M>& I)
{
  const int numThreadsPerBlock = 24;
  const dim3 threadsPerBlock = dim3(numThreadsPerBlock,numThreadsPerBlock);
  const dim3 numBlocks = dim3((O.width+threadsPerBlock.x)/threadsPerBlock.x,
                              (O.height+threadsPerBlock.y)/threadsPerBlock.y);

  krnlResampleBilinear<<<numBlocks,threadsPerBlock>>>(O,I);

  checkCudaError(hipDeviceSynchronize());
}

template<int NS,int NG,typename T>
struct PatchSSD_Split
{
  const TexArray2<NS,T> targetStyle;
  const TexArray2<NS,T> sourceStyle;

  const TexArray2<NG,T> targetGuide;
  const TexArray2<NG,T> sourceGuide;

  const Vec<NS,float> styleWeights;
  const Vec<NG,float> guideWeights;

  PatchSSD_Split(const TexArray2<NS,T>& targetStyle,
                 const TexArray2<NS,T>& sourceStyle,

                 const TexArray2<NG,T>& targetGuide,
                 const TexArray2<NG,T>& sourceGuide,

                 const Vec<NS,float>&   styleWeights,
                 const Vec<NG,float>&   guideWeights)

  : targetStyle(targetStyle),sourceStyle(sourceStyle),
    targetGuide(targetGuide),sourceGuide(sourceGuide),
    styleWeights(styleWeights),guideWeights(guideWeights) {}

   __device__ float operator()(const int   patchSize,
                               const int   tx,
                               const int   ty,
                               const int   sx,
                               const int   sy,
                               const float ebest)
  {
    const int r = patchSize/2;
    float error = 0;

    for(int py=-r;py<=+r;py++)
    {
      for(int px=-r;px<=+r;px++)
      {
        {
          const Vec<NS,T> pixTs = targetStyle(tx + px,ty + py);
          const Vec<NS,T> pixSs = sourceStyle(sx + px,sy + py);
          for(int i=0;i<NS;i++)
          {
            const float diff = float(pixTs[i]) - float(pixSs[i]);
            error += styleWeights[i]*diff*diff;
          }
        }

        {
          const Vec<NG,T> pixTg = targetGuide(tx + px,ty + py);
          const Vec<NG,T> pixSg = sourceGuide(sx + px,sy + py);
          for(int i=0;i<NG;i++)
          {
            const float diff = float(pixTg[i]) - float(pixSg[i]);
            error += guideWeights[i]*diff*diff;
          }
        }
      }

      if (error>ebest) { return error; }
    }

    return error;
  }
};

template<int NS,int NG,typename T>
struct PatchSSD_Split_Modulation
{
  const TexArray2<NS,T> targetStyle;
  const TexArray2<NS,T> sourceStyle;

  const TexArray2<NG,T> targetGuide;
  const TexArray2<NG,T> sourceGuide;

  const TexArray2<NG,unsigned char> targetModulation;

  const Vec<NS,float> styleWeights;
  const Vec<NG,float> guideWeights;

  PatchSSD_Split_Modulation(const TexArray2<NS,T>& targetStyle,
                            const TexArray2<NS,T>& sourceStyle,

                            const TexArray2<NG,T>& targetGuide,
                            const TexArray2<NG,T>& sourceGuide,

                            const TexArray2<NG,unsigned char>& targetModulation,

                            const Vec<NS,float>&   styleWeights,
                            const Vec<NG,float>&   guideWeights)

  : targetStyle(targetStyle),sourceStyle(sourceStyle),
    targetGuide(targetGuide),sourceGuide(sourceGuide),
    targetModulation(targetModulation),
    styleWeights(styleWeights),guideWeights(guideWeights) {}

   __device__ float operator()(const int   patchSize,
                               const int   tx,
                               const int   ty,
                               const int   sx,
                               const int   sy,
                               const float ebest)
  {
    const int r = patchSize/2;
    float error = 0;

    for(int py=-r;py<=+r;py++)
    {
      for(int px=-r;px<=+r;px++)
      {
        {
          const Vec<NS,T> pixTs = targetStyle(tx + px,ty + py);
          const Vec<NS,T> pixSs = sourceStyle(sx + px,sy + py);
          for(int i=0;i<NS;i++)
          {
            const float diff = float(pixTs[i]) - float(pixSs[i]);
            error += styleWeights[i]*diff*diff;
          }
        }

        {
          const Vec<NG,T> pixTg = targetGuide(tx + px,ty + py);
          const Vec<NG,T> pixSg = sourceGuide(sx + px,sy + py);
          const Vec<NG,float> mult = Vec<NG,float>(targetModulation(tx,ty))/255.0f;

          for(int i=0;i<NG;i++)
          {
            const float diff = float(pixTg[i]) - float(pixSg[i]);
            error += guideWeights[i]*mult[i]*diff*diff;
          }
        }
      }

      if (error>ebest) { return error; }
    }

    return error;
  }
};

V2i pyramidLevelSize(const V2i& sizeBase,const int numLevels,const int level)
{
  return V2i(V2f(sizeBase)*pow(2.0f,-float(numLevels-1-level)));
}

template<int NS,int NG>
void runEbsynth(int    ebsynthBackend,
                int    numStyleChannels,
                int    numGuideChannels,
                int    sourceWidth,
                int    sourceHeight,
                void*  sourceStyleData,
                void*  sourceGuideData,
                int    targetWidth,
                int    targetHeight,
                void*  targetGuideData,
                void*  targetModulationData,
                float* styleWeights,
                float* guideWeights,
                float  uniformityWeight,
                int    patchSize,
                int    voteMode,
                int    numPyramidLevels,
                int*   numSearchVoteItersPerLevel,
                int*   numPatchMatchItersPerLevel,
                int*   stopThresholdPerLevel,
                void*  outputData)
{
  const int levelCount = numPyramidLevels;

  struct PyramidLevel
  {
    PyramidLevel() { }

    int sourceWidth;
    int sourceHeight;
    int targetWidth;
    int targetHeight;

    TexArray2<NS,float> sourceStyle;
    TexArray2<NG,float> sourceGuide;
    TexArray2<NS,float> targetStyle;
    TexArray2<NS,float> targetStyle2;
    TexArray2<1,unsigned char>  mask;
    TexArray2<1,unsigned char>  mask2;
    TexArray2<NG,float> targetGuide;
    TexArray2<NG,unsigned char> targetModulation;
    TexArray2<2,int>            NNF;
    TexArray2<2,int>            NNF2;
    TexArray2<1,float>          E;
    MemArray2<int>              Omega;
  };

  std::vector<PyramidLevel> pyramid(levelCount);
  for(int level=0;level<levelCount;level++)
  {
    const V2i levelSourceSize = pyramidLevelSize(V2i(sourceWidth,sourceHeight),levelCount,level);
    const V2i levelTargetSize = pyramidLevelSize(V2i(targetWidth,targetHeight),levelCount,level);

    pyramid[level].sourceWidth  = levelSourceSize(0);
    pyramid[level].sourceHeight = levelSourceSize(1);
    pyramid[level].targetWidth  = levelTargetSize(0);
    pyramid[level].targetHeight = levelTargetSize(1);

    pyramid[level].sourceStyle  = TexArray2<NS,float>(levelSourceSize);
    pyramid[level].sourceGuide  = TexArray2<NG,float>(levelSourceSize);
    pyramid[level].targetStyle  = TexArray2<NS,float>(levelTargetSize);
    pyramid[level].targetStyle2 = TexArray2<NS,float>(levelTargetSize);
    pyramid[level].mask         = TexArray2<1,unsigned char>(levelTargetSize);
    pyramid[level].mask2        = TexArray2<1,unsigned char>(levelTargetSize);
    pyramid[level].targetGuide  = TexArray2<NG,float>(levelTargetSize);
    pyramid[level].NNF          = TexArray2<2,int>  (levelTargetSize);
    pyramid[level].NNF2         = TexArray2<2,int>  (levelTargetSize);
    pyramid[level].E            = TexArray2<1,float>(levelTargetSize);
    pyramid[level].Omega        = MemArray2<int>    (levelSourceSize);

    if (targetModulationData) { pyramid[level].targetModulation = TexArray2<NG,unsigned char>(levelTargetSize); }
  }

  copy(&pyramid[levelCount-1].sourceStyle,sourceStyleData);
  copy(&pyramid[levelCount-1].sourceGuide,sourceGuideData);
  copy(&pyramid[levelCount-1].targetGuide,targetGuideData);
  if (targetModulationData) { copy(&pyramid[levelCount-1].targetModulation,targetModulationData); }

  for(int level=0;level<levelCount-1;level++)
  {
    resampleGPU(pyramid[level].sourceStyle,pyramid[levelCount-1].sourceStyle);
    resampleGPU(pyramid[level].sourceGuide,pyramid[levelCount-1].sourceGuide);
    resampleGPU(pyramid[level].targetGuide,pyramid[levelCount-1].targetGuide);
    if (targetModulationData) { resampleGPU(pyramid[level].targetModulation,pyramid[levelCount-1].targetModulation); }
  }

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  bool inExtraPass = false;

  hiprandState* rngStates = initGpuRng(targetWidth,targetHeight);

  for (int level=0;level<pyramid.size();level++)
  {
    /////////////////////////////////////////////////////////////////////////////

    if (!inExtraPass)
    {
      A2V2i cpu_NNF;
      if (level>0)
      {
        A2V2i prevLevelNNF(pyramid[level-1].targetWidth,
                           pyramid[level-1].targetHeight);

        copy(&prevLevelNNF,pyramid[level-1].NNF);

        cpu_NNF = nnfUpscale(prevLevelNNF,
                             patchSize,
                             V2i(pyramid[level].targetWidth,pyramid[level].targetHeight),
                             V2i(pyramid[level].sourceWidth,pyramid[level].sourceHeight));
      }
      else
      {
        cpu_NNF = nnfInitRandom(V2i(pyramid[level].targetWidth,pyramid[level].targetHeight),
                                V2i(pyramid[level].sourceWidth,pyramid[level].sourceHeight),
                                patchSize);
      }
      copy(&pyramid[level].NNF,cpu_NNF);

      /////////////////////////////////////////////////////////////////////////
      Array2<int> cpu_Omega(pyramid[level].sourceWidth,pyramid[level].sourceHeight);

      fill(&cpu_Omega,(int)0);
      for(int ay=0;ay<cpu_NNF.height();ay++)
      for(int ax=0;ax<cpu_NNF.width();ax++)
      {
        const V2i& n = cpu_NNF(ax,ay);
        const int bx = n(0);
        const int by = n(1);

        const int r = patchSize/2;

        for(int oy=-r;oy<=+r;oy++)
        for(int ox=-r;ox<=+r;ox++)
        {
          const int x = bx+ox;
          const int y = by+oy;
          cpu_Omega(x,y) += 1;
        }
      }

      copy(&pyramid[level].Omega,cpu_Omega);
      /////////////////////////////////////////////////////////////////////////
    }

    ////////////////////////////////////////////////////////////////////////////
    {
      const int numThreadsPerBlock = 24;
      const dim3 threadsPerBlock = dim3(numThreadsPerBlock,numThreadsPerBlock);
      const dim3 numBlocks = dim3((pyramid[level].targetWidth+threadsPerBlock.x)/threadsPerBlock.x,
                                  (pyramid[level].targetHeight+threadsPerBlock.y)/threadsPerBlock.y);

      krnlVotePlain<<<numBlocks,threadsPerBlock>>>(pyramid[level].targetStyle2,
                                                   pyramid[level].sourceStyle,
                                                   pyramid[level].NNF,
                                                   patchSize);

      std::swap(pyramid[level].targetStyle2,pyramid[level].targetStyle);
      checkCudaError( hipDeviceSynchronize() );
    }
    ////////////////////////////////////////////////////////////////////////////

    Array2<Vec<1,unsigned char>> cpu_mask(V2i(pyramid[level].targetWidth,pyramid[level].targetHeight));
    fill(&cpu_mask,Vec<1,unsigned char>(255));
    copy(&pyramid[level].mask,cpu_mask);

    ////////////////////////////////////////////////////////////////////////////

    for (int voteIter=0;voteIter<numSearchVoteItersPerLevel[level];voteIter++)
    {
      Vec<NS,float> styleWeightsVec;
      for(int i=0;i<NS;i++) { styleWeightsVec[i] = styleWeights[i]; }

      Vec<NG,float> guideWeightsVec;
      for(int i=0;i<NG;i++) { guideWeightsVec[i] = guideWeights[i]; }

      const int numGpuThreadsPerBlock = 24;

      if (numPatchMatchItersPerLevel[level]>0)
      {
        if (targetModulationData)
        {
          patchmatchGPU(V2i(pyramid[level].targetWidth,pyramid[level].targetHeight),
                        V2i(pyramid[level].sourceWidth,pyramid[level].sourceHeight),
                        pyramid[level].Omega,
                        patchSize,
                        PatchSSD_Split_Modulation<NS,NG,float>(pyramid[level].targetStyle,
                                                                       pyramid[level].sourceStyle,
                                                                       pyramid[level].targetGuide,
                                                                       pyramid[level].sourceGuide,
                                                                       pyramid[level].targetModulation,
                                                                       styleWeightsVec,
                                                                       guideWeightsVec),
                        uniformityWeight,
                        numPatchMatchItersPerLevel[level],
                        numGpuThreadsPerBlock,
                        pyramid[level].NNF,
                        pyramid[level].NNF2,
                        pyramid[level].E,
                        pyramid[level].mask,
                        rngStates);
        }
        else
        {
          patchmatchGPU(V2i(pyramid[level].targetWidth,pyramid[level].targetHeight),
                        V2i(pyramid[level].sourceWidth,pyramid[level].sourceHeight),
                        pyramid[level].Omega,
                        patchSize,
                        PatchSSD_Split<NS,NG,float>(pyramid[level].targetStyle,
                                                            pyramid[level].sourceStyle,
                                                            pyramid[level].targetGuide,
                                                            pyramid[level].sourceGuide,
                                                            styleWeightsVec,
                                                            guideWeightsVec),
                        uniformityWeight,
                        numPatchMatchItersPerLevel[level],
                        numGpuThreadsPerBlock,
                        pyramid[level].NNF,
                        pyramid[level].NNF2,
                        pyramid[level].E,
                        pyramid[level].mask,
                        rngStates);
        }
      }
      else
      {
        const int numThreadsPerBlock = 24;
        const dim3 threadsPerBlock = dim3(numThreadsPerBlock,numThreadsPerBlock);
        const dim3 numBlocks = dim3((pyramid[level].targetWidth+threadsPerBlock.x)/threadsPerBlock.x,
                                    (pyramid[level].targetHeight+threadsPerBlock.y)/threadsPerBlock.y);

        if (targetModulationData)
        {
          krnlEvalErrorPass<<<numBlocks,threadsPerBlock>>>(patchSize,
                                                           PatchSSD_Split_Modulation<NS,NG,float>(pyramid[level].targetStyle,
                                                                                                  pyramid[level].sourceStyle,
                                                                                                  pyramid[level].targetGuide,
                                                                                                  pyramid[level].sourceGuide,
                                                                                                  pyramid[level].targetModulation,
                                                                                                  styleWeightsVec,
                                                                                                  guideWeightsVec),
                                                           pyramid[level].NNF,
                                                           pyramid[level].E);
        }
        else
        {
          krnlEvalErrorPass<<<numBlocks,threadsPerBlock>>>(patchSize,
                                                           PatchSSD_Split<NS,NG,float>(pyramid[level].targetStyle,
                                                                                       pyramid[level].sourceStyle,
                                                                                       pyramid[level].targetGuide,
                                                                                       pyramid[level].sourceGuide,
                                                                                       styleWeightsVec,
                                                                                       guideWeightsVec),
                                                           pyramid[level].NNF,
                                                           pyramid[level].E);
        }
        checkCudaError( hipDeviceSynchronize() );
      }

      {
        const int numThreadsPerBlock = 24;
        const dim3 threadsPerBlock = dim3(numThreadsPerBlock,numThreadsPerBlock);
        const dim3 numBlocks = dim3((pyramid[level].targetWidth+threadsPerBlock.x)/threadsPerBlock.x,
                                    (pyramid[level].targetHeight+threadsPerBlock.y)/threadsPerBlock.y);

        if      (voteMode==EBSYNTH_VOTEMODE_PLAIN)
        {
          krnlVotePlain<<<numBlocks,threadsPerBlock>>>(pyramid[level].targetStyle2,
                                                       pyramid[level].sourceStyle,
                                                       pyramid[level].NNF,
                                                       patchSize);
        }
        else if (voteMode==EBSYNTH_VOTEMODE_WEIGHTED)
        {
          krnlVoteWeighted<<<numBlocks,threadsPerBlock>>>(pyramid[level].targetStyle2,
                                                          pyramid[level].sourceStyle,
                                                          pyramid[level].NNF,
                                                          pyramid[level].E,
                                                          patchSize);
        }

        std::swap(pyramid[level].targetStyle2,pyramid[level].targetStyle);
        checkCudaError( hipDeviceSynchronize() );

        if (voteIter<numSearchVoteItersPerLevel[level]-1)
        {
          krnlEvalMask<<<numBlocks,threadsPerBlock>>>(pyramid[level].mask,
                                                      pyramid[level].targetStyle,
                                                      pyramid[level].targetStyle2,
                                                      stopThresholdPerLevel[level]);
          checkCudaError( hipDeviceSynchronize() );

          krnlDilateMask<<<numBlocks,threadsPerBlock>>>(pyramid[level].mask2,
                                                        pyramid[level].mask,
                                                        patchSize);
          std::swap(pyramid[level].mask2,pyramid[level].mask);
          checkCudaError( hipDeviceSynchronize() );
        }
      }
    }
  }

  checkCudaError( hipDeviceSynchronize() );

  copy(&outputData,pyramid[pyramid.size()-1].targetStyle);

  checkCudaError( hipFree(rngStates) );

  for(int level=0;level<pyramid.size();level++)
  {
    pyramid[level].sourceStyle.destroy();
    pyramid[level].sourceGuide.destroy();
    pyramid[level].targetStyle.destroy();
    pyramid[level].targetStyle2.destroy();
    pyramid[level].mask.destroy();
    pyramid[level].mask2.destroy();
    pyramid[level].targetGuide.destroy();
    pyramid[level].NNF.destroy();
    pyramid[level].NNF2.destroy();
    pyramid[level].E.destroy();
    pyramid[level].Omega.destroy();
    if (targetModulationData) { pyramid[level].targetModulation.destroy(); }
  }
}

EBSYNTH_API void ebsynthRun(int    ebsynthBackend,
                            int    numStyleChannels,
                            int    numGuideChannels,
                            int    sourceWidth,
                            int    sourceHeight,
                            void*  sourceStyleData,
                            void*  sourceGuideData,
                            int    targetWidth,
                            int    targetHeight,
                            void*  targetGuideData,
                            void*  targetModulationData,
                            float* styleWeights,
                            float* guideWeights,
                            float  uniformityWeight,
                            int    patchSize,
                            int    voteMode,
                            int    numPyramidLevels,
                            int*   numSearchVoteItersPerLevel,
                            int*   numPatchMatchItersPerLevel,
                            int*   stopThresholdPerLevel,
                            void*  outputData
                            )
{
  void(*const dispatchEbsynth[EBSYNTH_MAX_GUIDE_CHANNELS])(int, int, int, int, int, void*, void*, int, int, void*, void*, float*, float*, float, int, int, int, int*, int*, int*, void*) =
  {
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,1>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,2>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,3>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,4>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,5>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,6>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,7>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,8>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,9>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,10>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,11>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,12>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,13>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,14>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,15>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,16>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,17>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,18>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,19>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,20>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,21>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,22>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,23>,
	  runEbsynth<EBSYNTH_NUM_STYLE_CHANNELS,24>,
  };

  if (numStyleChannels != EBSYNTH_NUM_STYLE_CHANNELS) { printf("ebsynth.dll error: this version only supports exactly %d style channels!\n", EBSYNTH_NUM_STYLE_CHANNELS); return; }
  if (numGuideChannels<1) { printf("ebsynth.dll error: expecting at least one guide channel!\n"); return; }
  if (numGuideChannels>EBSYNTH_MAX_GUIDE_CHANNELS) { printf("ebsynth.dll error: too many guide channels!\n"); return; }

  if (numGuideChannels>=1 && numGuideChannels<=EBSYNTH_MAX_GUIDE_CHANNELS)
  {
    dispatchEbsynth[numGuideChannels-1](ebsynthBackend,
                                        numStyleChannels,
                                        numGuideChannels,
                                        sourceWidth,
                                        sourceHeight,
                                        sourceStyleData,
                                        sourceGuideData,
                                        targetWidth,
                                        targetHeight,
                                        targetGuideData,
                                        targetModulationData,
                                        styleWeights,
                                        guideWeights,
                                        uniformityWeight,
                                        patchSize,
                                        voteMode,
                                        numPyramidLevels,
                                        numSearchVoteItersPerLevel,
                                        numPatchMatchItersPerLevel,
                                        stopThresholdPerLevel,
                                        outputData);
  }
}

EBSYNTH_API
int ebsynthBackendAvailable(int ebsynthBackend)
{
  if (ebsynthBackend==EBSYNTH_BACKEND_CUDA)
  {
    int deviceCount = -1;
    if (hipGetDeviceCount(&deviceCount)!=hipSuccess) { return 0; }

    for (int device=0;device<deviceCount;device++)
    {
      hipDeviceProp_t properties;
      if (hipGetDeviceProperties(&properties,device)==hipSuccess)
      {
        if (properties.major!=9999 && properties.major>=3)
        {
          return 1;
        }
      }
    }
  }

  return 0;
}
